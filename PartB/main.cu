#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <ctime>
#include <chrono>
#include <fstream>
#include <assert.h>
using namespace std;

#include "gpu_thread.h"

// Used to cross-check answer. DO NOT MODIFY!
void reference(int N, int *matA, int *matB, int *output)
{
    // enforce N to be power of 2 and greater than 2
    assert( N>=4 and N == ( N &~ (N-1)));
    for(int rowA = 0; rowA < N; rowA++){
        for(int colSetB = 0; colSetB < N; colSetB += 2){
            int sum = 0;
            if(rowA & 1) {    
                // handle odd rows in matA
                
                // iterate over even positions in rowA
                // and odd positions in column colSetB in matB
                for(int indexA = rowA*N, indexB = colSetB+N; indexA < (rowA+1)*N; 
                        indexA += 2, indexB += (N<<1))
                    sum += matA[indexA] * matB[indexB];
                
                // iterate over odd positions in rowA
                // and even positions in column colSetB+1 in matB
                for(int indexA = rowA*N+1, indexB = colSetB+1; indexA < (rowA+1)*N; 
                        indexA += 2, indexB += (N<<1))
                    sum += matA[indexA] * matB[indexB];
            
            } else {
                // handle even rows in matA

                // iterate over even positions in rowA
                // and odd positions in column colSetB+1 in matB
                for(int indexA = rowA*N, indexB = colSetB+1+N; indexA < (rowA+1)*N; 
                        indexA += 2, indexB += (N<<1))
                    sum += matA[indexA] * matB[indexB];

                // iterate over odd positions in rowA
                // and even positions in column colSetB in matB
                for(int indexA = rowA*N+1, indexB = colSetB; indexA < (rowA+1)*N; 
                        indexA += 2, indexB += (N<<1))
                    sum += matA[indexA] * matB[indexB];
            }
            
            // compute output indices
            int rowC = rowA>>1;
            int colC = (colSetB>>1) + (N>>1) * (rowA&1);
            int indexC = rowC * N + colC;
            output[indexC] = sum;
        }
    }

}

int main(int argc, char *argv[])
{
    // Input size of square matrices// Input size of square matrices
    int N;
    string file_name; 
    if (argc < 2) 
        file_name = "data/input_128.in"; 
    else 
        file_name = argv[1]; 
    ifstream input_file; 
    input_file.open(file_name); 
    input_file >> N;
    cout << "Input matrix of size " << N << "\n";
    
    // Input matrix A
    int *matA = new int[N * N];
    for(int i = 0; i < N; ++i)
        for(int j = 0; j < N; ++j)
            input_file >> matA[i * N + j];

    // Input matrix B
    int *matB = new int[N * N];
    for(int i = 0; i < N; ++i)
        for(int j = 0; j < N; ++j)
            input_file >> matB[i * N + j];
    
    // Execute reference program
    int *output_reference = new int[N*(N>>1)];
    reference(N, matA, matB, output_reference);

    // Execute gpu version
    int *output_gpu = new int[N*(N>>1)];
    gpuThread(N, matA, matB, output_gpu);
    for(int i = 0; i < N*(N>>1); ++i)
        if(output_gpu[i] != output_reference[i]) {
            cout << "Mismatch at " << i << "\n";
            cout << "GPU output: " << output_gpu[i] << ", required output: " << output_reference[i] << "\n";
            exit(0);
        }
    cout<<"Everything is fine"<<endl;
    input_file.close(); 
}
